#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

#define	R2 2
#define	R4 4
#define	R8 8


__global__ void KernelFunc1(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M, const unsigned int Ns) 
{	
	int j = bx * blockDim.x + tx;
	int idxS = j;
	float v_r[R2]; float v_i[R2];
	float angle = -2*PI*(j%Ns) / (Ns*R2);
	for (int r=0; r<R2; r++)
	{	
		v_r[r] = (x_r_d[idxS+r*N/R2]*cos(r*angle) - x_i_d[idxS+r*N/R2]*sin(r*angle));
		v_i[r] = (x_r_d[idxS+r*N/R2]*sin(r*angle) + x_i_d[idxS+r*N/R2]*cos(r*angle));			
	}
	
	float v0_r = v_r[0]; float v0_i = v_i[0];
	v_r[0] = v0_r + v_r[1]; v_i[0] = v0_i + v_i[1];
	v_r[1] = v0_r - v_r[1]; v_i[1] = v0_i - v_i[1];
	
	int idxD = (j/Ns)*Ns*R2 + (j%Ns);
	
	for (int r=0; r<R2; r++)
	{
		X_r_d[idxD + r*Ns] = v_r[r];
		X_i_d[idxD + r*Ns] = v_i[r];
	}
}

__global__ void KernelFunc2(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M, const unsigned int Ns) 
{	
	int j = bx * blockDim.x + tx;
	int idxS = j;
	float v_r[R4]; float v_i[R4];
	float angle = -2*PI*(j%Ns) / (Ns*R4);
	for (int r=0; r<R4; r++)
	{		
		v_r[r] = x_r_d[idxS+r*N/R4]*cos(r*angle) - x_i_d[idxS+r*N/R4]*sin(r*angle);
		v_i[r] = x_r_d[idxS+r*N/R4]*sin(r*angle) + x_i_d[idxS+r*N/R4]*cos(r*angle);			
	}

	float v0_r = v_r[0]; float v0_i = v_i[0];
	float v1_r = v_r[1]; float v1_i = v_i[1];
	float v2_r = v_r[2]; float v2_i = v_i[2];
	float v3_r = v_r[3]; float v3_i = v_i[3];
	v_r[0] = v0_r + v1_r + v2_r + v3_r; 	v_i[0] = v0_i + v1_i + v2_i + v3_i;
	v_r[1] = v0_r + v1_i - v2_r - v3_i;		v_i[1] = v0_i - v1_r - v2_i + v3_r;
	v_r[2] = v0_r - v1_r + v2_r - v3_r;		v_i[2] = v0_i - v1_i + v2_i - v3_i;
	v_r[3] = v0_r - v1_i - v2_r + v3_i;		v_i[3] = v0_i + v1_r - v2_i - v3_r;
	
	int idxD = (j/Ns)*Ns*R4 + (j%Ns);
	for (int r=0; r<R4 ; r++)
	{
		X_r_d[idxD + r*Ns] = v_r[r];
		X_i_d[idxD + r*Ns] = v_i[r];
	}
}

__global__ void KernelFunc3(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M, const unsigned int Ns) 
{	
	int j = bx * blockDim.x + tx;
	int idxS = j;
	float v_r[R8]; float v_i[R8];
	float angle = -2*PI*(j%Ns) / (Ns*R8);
	for (int r=0; r<R8; r++)
	{		
		v_r[r] = x_r_d[idxS+r*N/R8]*cos(r*angle) - x_i_d[idxS+r*N/R8]*sin(r*angle);
		v_i[r] = x_r_d[idxS+r*N/R8]*sin(r*angle) + x_i_d[idxS+r*N/R8]*cos(r*angle);			
	}

	float a= 0.7071;
	float v0_r = v_r[0]; float v0_i = v_i[0];
	float v1_r = v_r[1]; float v1_i = v_i[1];
	float v2_r = v_r[2]; float v2_i = v_i[2];
	float v3_r = v_r[3]; float v3_i = v_i[3];
	float v4_r = v_r[4]; float v4_i = v_i[4];
	float v5_r = v_r[5]; float v5_i = v_i[5];
	float v6_r = v_r[6]; float v6_i = v_i[6];
	float v7_r = v_r[7]; float v7_i = v_i[7];
	v_r[0] = v0_r + v1_r + v2_r + v3_r + v4_r + v5_r + v6_r + v7_r;
	v_i[0] = v0_i + v1_i + v2_i + v3_i + v4_i + v5_i + v6_i + v7_i;
	
	v_r[1] = v0_r + a*(v1_r+v1_i) + v2_i + a*(v3_i-v3_r) + v4_r - a*(v5_r+v5_i) - v6_i + a*(v7_r-v7_i);
	v_i[1] = v0_i + a*(v1_i-v1_r) - v2_r - a*(v3_r+v3_i) + v4_i + a*(v5_r-v5_i) + v6_r + a*(v7_r+v7_i);
	
	v_r[2] = v0_r + v1_i + v2_r - v3_i + v4_r + v5_i - v6_r - v7_i;
	v_i[2] = v0_i - v1_r + v2_i + v3_r + v4_i - v5_r - v6_i + v7_r;
	
	v_r[3] = v0_r + a*(v1_i-v1_r) - v2_i + a*(v3_r+v3_i) + v4_r - a*(v5_i-v5_r) + v6_i - a*(v7_r+v7_i);
	v_i[3] = v0_i - a*(v1_r+v1_i) + v2_r + a*(v3_i-v3_r) + v4_i + a*(v5_r+v5_i) - v6_r + a*(v7_r-v7_i);
	
	v_r[4] = v0_r - v1_r + v2_r - v3_r + v4_r - v5_r + v6_r - v7_r;
	v_i[4] = v0_i - v1_i + v2_i - v3_i + v4_i - v5_i + v6_i - v7_i;
	
	v_r[5] = v0_r - a*(v1_r+v1_i) + v2_i - a*(v3_i-v3_r) + v4_r + a*(v5_r+v5_i) - v6_i - a*(v7_r-v7_i);
	v_i[5] = v0_i - a*(v1_i-v1_r) - v2_r + a*(v3_r+v3_i) + v4_i - a*(v5_r-v5_i) + v6_r - a*(v7_r+v7_i);
	
	v_r[6] = v0_r - v1_i + v2_r + v3_i + v4_r - v5_i - v6_r + v7_i;
	v_i[6] = v0_i + v1_r + v2_i - v3_r + v4_i + v5_r - v6_i - v7_r;
	
	v_r[7] = v0_r - a*(v1_i-v1_r) - v2_i - a*(v3_r+v3_i) + v4_r + a*(v5_i-v5_r) + v6_i + a*(v7_r+v7_i);
	v_i[7] = v0_i + a*(v1_r+v1_i) + v2_r - a*(v3_i-v3_r) + v4_i - a*(v5_r+v5_i) - v6_r - a*(v7_r-v7_i);
	
	int idxD = (j/Ns)*Ns*R8 + (j%Ns);
	for (int r=0; r<R8 ; r++)
	{
		X_r_d[idxD + r*Ns] = v_r[r];
		X_i_d[idxD + r*Ns] = v_i[r];
	}
}

__global__ void KernelCopy (float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d)
{	
	int t_Id = bx * blockDim.x + tx;
	x_r_d[t_Id] = X_r_d[t_Id];
	x_i_d[t_Id] = X_i_d[t_Id];	
}


//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, /*float* X_r_d, float* X_i_d,*/ const unsigned int N, const unsigned int M)
{
	if(M>25)
	{
		gpuKernel(x_r_d,x_i_d,N/2,M-1);
		gpuKernel(&x_r_d[N/2],&x_i_d[N/2],N/2,M-1);
		
		return;
	}
	
	float* X_r_d;
	float* X_i_d;	
	HANDLE_ERROR(hipMalloc((void**)&X_r_d, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&X_i_d, N * sizeof(float)));
	
	int Ns;
	if ((M%4==0) && M>10)
	{
		for (Ns=1; Ns<N ; Ns*=R4)
		{	
			KernelFunc2 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
			Ns = Ns*R4;
			KernelFunc2 <<< N/(1024*R4), 1024 >>>(X_r_d, X_i_d, x_r_d, x_i_d, N, M, Ns);
		}
	}
	else if(M==11)
	{
		for (Ns=1; Ns<N ; Ns*=R2)
			{							
				KernelFunc1 <<< 1, 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
				KernelCopy <<< 2, 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}
	}
	else if (M>10)
	{
		if (M%2)
		{
			for (Ns=1; Ns<(N/2) ; Ns*=R4)
			{							
				KernelFunc2 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
				KernelCopy <<< (1<<M-10), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}
			KernelFunc1 <<< N/(1024*R2), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
			KernelCopy <<< (1<<M-10), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d);
		}
		else
			for (Ns=1; Ns<N ; Ns*=R4)
			{							
				KernelFunc2 <<< N/(1024*R4), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
				KernelCopy <<< (1<<M-10), 1024 >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}		
	}
	else 
	{
		if (M%2)
		{
			for (Ns=1; Ns<(N/2) ; Ns*=R4)
			{							
				KernelFunc2 <<< 1, N/R4 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
				KernelCopy <<< 1, N >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}
			KernelFunc1 <<< 1, N/R2 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
			KernelCopy <<< 1, N >>>(x_r_d, x_i_d, X_r_d, X_i_d);
		}
		else
			for (Ns=1; Ns<N ; Ns*=R4)
			{							
				KernelFunc2 <<< 1, N/R4 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, Ns);
				KernelCopy <<< 1, N >>>(x_r_d, x_i_d, X_r_d, X_i_d);
			}		
	}
	
    HANDLE_ERROR(hipFree(X_r_d));
    HANDLE_ERROR(hipFree(X_i_d));
}
