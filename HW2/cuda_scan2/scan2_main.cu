#include "hip/hip_runtime.h"
//Do NOT MODIFY THIS FILE
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#include "gpuerrors.h"
#include "scan2.h"

//-----------------------------------------------------------------------------
void fill(float* data, int size) {			// to fill array with values in range [-2,+2]
    for (int i=0; i<size; ++i)
        data[i] = (float) 1.0*((rand() % 5)-2);
}

double calc_mse (float* data1, float* data2, int size) {	// calculate error by (  )^2
	double mse = 0.0;
	int i; 
	double e = 0.0;
	for (i=0; i<size; i++) {
		e = data1[i] - data2[i];
		e = e * e;
		mse += e;
	}
	mse = mse / ((double)size);
	return mse;
}
//-----------------------------------------------------------------------------
void cpuKernel (float* a, float* c, int n) {		// calculate scan algorithm  
	int i = 0;
	float sum = 0;
	for (i = 0; i < n; i += 1){
		//c[i] = sum;			// to have exclusive scan
		sum += (a[i]);
		c[i] = sum;			// to have inclusive scan
	}
	return;
}
//-----------------------------------------------------------------------------
int main ( int argc, char** argv) {
	   
	float* a;
	float* c_serial;
	float* c;	
	
	int m = 5; 
	int n = 32;

	if (argc > 1){
		m = atoi(argv[1]);
		n = (1 << m);
	}

	a        = (float*)malloc(n * sizeof(float));
	c_serial = (float*)malloc(n * sizeof(float));
	c        = (float*)malloc(n * sizeof(float));
				
	srand(0);
	fill(a, n);

	cpuKernel (a, c_serial, n);
		
	clock_t t0 = clock(); 
	gpuKernel (a, c, n);
	clock_t t1 = clock();
		
	float mse;
	mse = calc_mse( c_serial, c, n );
	
	printf("n=%d\t time=%06ld ms mse=%f\n",n, (t1-t0)/1000, mse);	
	//printf("%d\t%d\t%d\t%d\t%d\t", c[0],c[1],c[2],c[3],c[4]);

	free(a);
	free(c_serial);
	free(c);
	return 0;
}
//Do NOT MODIFY THIS FILE

