#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE

#include "scan2.h"
#include "gpuerrors.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!


__global__ void kernelFunc(float* ad, float* cd, float* a1d, const int BlockSize,  int inclusive_EN) {
	__shared__ float ads[1024];
	int i = tx;
	int j = ((1<<25)/2)*by + (BlockSize/2)*bx + tx;

	ads[2*i] = ad[2*j];
	ads[2*i +1] = ad[2*j +1];
	//__syncthreads();

	for(int k=1; k<=BlockSize/2; k*=2) {
		//__syncthreads();
		if( i < BlockSize/(2*k) )
			ads[(k*2)*(i+1) -1] += ads[k*(2*i+1) -1];

		__syncthreads();
	}

	float sum;
	if(i == BlockSize/2 -1) {
		sum = ads[BlockSize -1];
		a1d[(1<<15)*by + bx] = sum;
	}
	__syncthreads();
	if(i==0)
		ads[BlockSize -1] = 0;

	float temp;
	for(int k=BlockSize/2; k>=1; k= k/2){
		//__syncthreads();
		if( i < BlockSize/(2*k) ){
			temp = ads[(k*2)*(i+1) -1];
			ads[(k*2)*(i+1) -1] += ads[k*(2*i+1) -1];
			ads[k*(2*i+1) -1] = temp;
		}
		__syncthreads();
	}

	if(inclusive_EN) {
		if(i == BlockSize/2 -1) {
			cd[2*j] = ads[2*i +1];
			cd[2*j +1] = sum;
		}
		else {
			cd[2*j] = ads[2*i +1];
			cd[2*j +1] = ads[2*i +2];
		}
	}
	else {
		cd[2*j] = ads[2*i];
		cd[2*j +1] = ads[2*i+1];
	}
}


__global__ void kernelFunc2(float* cd, float* ad, float OldSum=0.0) {
	cd[(1<<25)*by + 1024*bx + tx] += ad[(1<<15)*by + bx] + OldSum;
}


void gpuKernel(float* a, float* c,int n) {
	float* ad;
	float* a1d;
	float* a2d;
	float* cd;

    	HANDLE_ERROR(hipMalloc((void**)&cd, sizeof(float)));

			if(n < 1<<28) {
					HANDLE_ERROR(hipMalloc((void**)&ad, n * sizeof(float)));
		    	HANDLE_ERROR(hipMalloc((void**)&a1d, (n/1024) * sizeof(float)));
		    	HANDLE_ERROR(hipMalloc((void**)&a2d, (n/(1024*1024)) * sizeof(float)));

		    	HANDLE_ERROR(hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice));

					if(n == 1<<20) {
						kernelFunc <<< n/1024, 512 >>> (ad, ad, a1d, 1024, 1);
						kernelFunc <<< n/(1024*1024), 512 >>> (a1d, a1d, a2d, 1024, 0);
						kernelFunc2 <<< n/1024, 1024 >>> (ad, a1d);
					}
					else if(n < 1<<26) {
						kernelFunc <<< n/1024, 512 >>> (ad, ad, a1d, 1024, 1);
						kernelFunc <<< n/(1024*1024), 512 >>> (a1d, a1d, a2d, 1024, 0);
						kernelFunc <<< 1, 512 >>> (a2d, a2d, cd, 1024, 0);
						kernelFunc2 <<< n/(1024*1024), 1024 >>> (a1d, a2d);
						kernelFunc2 <<< n/1024, 1024 >>> (ad, a1d);
					}
					else {
						dim3 dimGrid(1<<15, n/(1<<25));
						kernelFunc <<< dimGrid, 512 >>> (ad, ad, a1d, 1024, 1);
						kernelFunc <<< n/(1024*1024), 512 >>> (a1d, a1d, a2d, 1024, 0);
						kernelFunc <<< 1, 512 >>> (a2d, a2d, cd, 1024, 0);
						kernelFunc2 <<< n/(1024*1024), 1024 >>> (a1d, a2d);
						kernelFunc2 <<< dimGrid, 1024 >>> (ad, a1d);
					}

					HANDLE_ERROR(hipMemcpy(c, ad, n * sizeof(float), hipMemcpyDeviceToHost));
			}

			else {
					int MaxMemSize = 1<<27;
					float OldSum=0;
					HANDLE_ERROR(hipMalloc((void**)&ad, MaxMemSize * sizeof(float)));
		    	HANDLE_ERROR(hipMalloc((void**)&a1d, (MaxMemSize/1024) * sizeof(float)));
		    	HANDLE_ERROR(hipMalloc((void**)&a2d, (MaxMemSize/(1024*1024)) * sizeof(float)));

					int NumberofIter = n/MaxMemSize;
					for(int i=0; i<NumberofIter; i++) {
						HANDLE_ERROR(hipMemcpy(ad, a + i*MaxMemSize, MaxMemSize * sizeof(float), hipMemcpyHostToDevice));

						dim3 dimGrid(1<<15, MaxMemSize/(1<<25));
						kernelFunc <<< dimGrid, 512 >>> (ad, ad, a1d, 1024, 1);
						kernelFunc <<< MaxMemSize/(1024*1024), 512 >>> (a1d, a1d, a2d, 1024, 0);
						kernelFunc <<< 1, 512 >>> (a2d, a2d, cd, 1024, 0);
						kernelFunc2 <<< MaxMemSize/(1024*1024), 1024 >>> (a1d, a2d);
						kernelFunc2 <<< dimGrid, 1024 >>> (ad, a1d, OldSum);

						HANDLE_ERROR(hipMemcpy(c + i*MaxMemSize, ad, MaxMemSize * sizeof(float), hipMemcpyDeviceToHost));
						OldSum = c[(i+1)*MaxMemSize-1];
					}
			}

    	HANDLE_ERROR(hipFree(ad));
			HANDLE_ERROR(hipFree(a1d));
			HANDLE_ERROR(hipFree(a2d));
			HANDLE_ERROR(hipFree(cd));
}
