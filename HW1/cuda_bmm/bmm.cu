#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 32
#define TILEY 16

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!
#define minTile (TILEX < TILEY ? TILEX : TILEY)

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {
	//const int T = ((TILEX/TILEY)>4 || (TILEY/TILEX)>4) ? 8*minTile : 4*minTile;
	const int T = ((TILEX==32) && (TILEY==32)) ? 4*minTile : 8*minTile;

	// write your GPU kernel function here
	__shared__ float ads[TILEY][T];
	__shared__ float bds[T][TILEX];
	
	int Row = by * TILEY + ty;
	int Col = bx * TILEX + tx;
	
	int lx = T/TILEX;
	int ly = T/TILEY;
	
	float sum = 0.0;

	for(int p=0; p<n/T; p++){
		for(int k = 0; k < lx; k++)
			ads[ty][tx + k*TILEX] = ad[Row*n + p*T + tx + k*TILEX];		

		for(int k = 0; k < ly; k++)
			bds[ty + k*TILEY][tx] = bd[(p*T + ty + k*TILEY)*n + Col];
		__syncthreads();

		for (int k = 0; k < T; k++)
			sum += ads[ty][k] * bds[k][tx];
		__syncthreads();
	}
	cd[Row*n + Col] = sum;
}

